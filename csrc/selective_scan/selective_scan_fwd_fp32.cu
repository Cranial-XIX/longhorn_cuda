/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_A_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_A_fwd_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_B_fwd_cuda<float, float>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_B_fwd_cuda<float, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_new_fwd_cuda<float, float>(SSMNewParamsBase &params, hipStream_t stream);
template void selective_scan_decay_fwd_cuda<float, float>(SSMDecayParamsBase &params, hipStream_t stream);
template void selective_scan_new_fwd_cuda<float, complex_t>(SSMNewParamsBase &params, hipStream_t stream);
