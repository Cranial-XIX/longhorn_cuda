/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_fwd_cuda<at::Half, complex_t>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_A_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_A_fwd_cuda<at::Half, complex_t>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_B_fwd_cuda<at::Half, float>(SSMParamsBase &params, hipStream_t stream);
// template void selective_scan_B_fwd_cuda<at::Half, complex_t>(SSMParamsBase &params, hipStream_t stream);
template void selective_scan_new_fwd_cuda<at::Half, float>(SSMNewParamsBase &params, hipStream_t stream);
template void selective_scan_decay_fwd_cuda<at::Half, float>(SSMDecayParamsBase &params, hipStream_t stream);
template void selective_scan_new_fwd_cuda<at::Half, complex_t>(SSMNewParamsBase &params, hipStream_t stream);
